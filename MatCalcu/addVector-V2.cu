#include "hip/hip_runtime.h"   // Unix系统下调用CUDA使用"cuda.h"，Win系统下调用CUDA使用"cuda.runtime"
#include "stdio.h"  // 标准输入输出，后面调用函数printf
#define  N	10000

__global__ void vectorAdd(float *A, float *B, float *C)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	C[i*N+j] = A[i*N+j] + B[i*N+j];
	printf("======================================================\n");
	printf("blockIdx.x:%d\tblockIdx.y:%d\n", blockIdx.x, blockIdx.y);
	printf("blockDim.x:%d\tblockDim.y:%d\n", blockDim.x, blockDim.y);
	printf("threadIdx.x:%d\tthreadIdx.y:%d\n", threadIdx.x, threadIdx.y);
	printf("i:%d j:%d C[i*N+j]:C[%2d]:%f\n", i, j, i*N+j, C[i*N+j]);
}

int main()
{
	// 设置使用的GPU下标(从0开始)
	// 多个GPU在终端用命令'nvidia-smi'查看下标
	int gpuDeviceIdx = 0;
	hipSetDevice(gpuDeviceIdx);

	// 初始化设备(CPU)端变量
	// 使用cudaMalloc方法分配指定大小的空间
	float *Ad, *Bd, *Cd;
	int size = N*N*sizeof(float);
	hipMalloc((void**)&Ad, size);
	hipMalloc((void**)&Bd, size);
	hipMalloc((void**)&Cd, size);

	// 设置程序在GPU上运行参数
	//	1.grid下的block数目
	//	2.block下的thread数目
	dim3 bpg(10, 10); // 每个网格下的线程块个数(block num. per grid)
					// bpg(blockDim.x, blockDim.y)
	dim3 tpb(10, 10); // 每个线程块下的线程个数(thread num. per block)
					// tpb(threadIdx.x, threadIdx.y)

	// 根据资源分配的参数
	// 在GPU上执行核函数(__global__修饰的函数)
	vectorAdd<<<bpg, tpb>>>(Ad, Bd, Cd);

	// 释放空间
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);

	return 0;
}
