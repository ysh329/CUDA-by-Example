#include "hip/hip_runtime.h"
#include "malloc.h"
#include "stdio.h"

#define N 4

// GPU端矩阵转置
__global__ void matrixTranspose(float *Ad, float *Bh, int rowElemNumInAd, int colElemNumInAd)
{
	int cCol = threadIdx.x; 
	int cRow = threadIdx.y;
	*(Bh+rowElemNumInAd*cCol+cRow) = *(Ad+colElemNumInAd*cRow+cCol);
}

// CPU端矩阵初始化
void matrixInitialize(float *Ah, int rowElemNum, int colElemNum)
{
	float summ = 0.0;
	int i, j;

    for (i = 0; i < rowElemNum; i++)
    {
        for (j = 0; j < colElemNum; j++)
        {   
			summ += 1.0;
			*(Ah+colElemNum*i+j) = summ; 
		}
    }
}

// CPU端打印矩阵
void matrixPrint(float *Ah, int rowElemNum, int colElemNum)
{
	int i, j;

	for (i = 0; i < rowElemNum; i++)
	{
		for (j = 0; j < colElemNum; j++)
		{
			printf("%2.2f \t", *(Ah+colElemNum*i+j));
		}
		printf("\n");
	}
}

// CPU端打印矩阵展平后对应的向量
__host__ void vectorPrint(float *Vh, int elemNum)
{
	int i;

	for (i = 0; i < elemNum; i++)
	{
		printf("%2.2f ", *(Vh+i));
	}
	printf("\n");
}

int main()
{
	// 变量初始化
	float *Ad, *Bd; // GPU设备端变量: Ad为原始矩阵, Bd为转置后矩阵
	float *Ah, *Bh; // CPU主机端变量: Bh为转置后矩阵

	// 设定原始矩阵A的维度
	int rowElemNumInA = N, colElemNumInA = N*rowElemNumInA; // dimNum = 2
	int rowElemNumInB = colElemNumInA, colElemNumInB = rowElemNumInA;
	int size = sizeof(float) * rowElemNumInA * colElemNumInA;

	// GPU参数初始化
    // 设置使用的GPU下标(从0开始)
    // 多个GPU在终端用命令'nvidia-smi'查看下标
    int gpuDeviceIdx = 0;
    hipSetDevice(gpuDeviceIdx);

	// CPU主机端申请空间
	Ah = (float*)malloc(size);
	Bh = (float*)malloc(size);

	// GPU设备端申请空间	
	hipMalloc((void**)&Ad, size);
	hipMalloc((void**)&Bd, size);

	// CPU端初始化原始矩阵A
	matrixInitialize(Ah, rowElemNumInA, colElemNumInA);

	// CPU端打印原始矩阵A
	matrixPrint(Ah, rowElemNumInA, colElemNumInA);

	// GPU端转置矩阵Ad后保存到矩阵Bh
    // 设置程序在GPU上运行参数
    //  1.grid下的block数目
    //  2.block下的thread数目
    dim3 bpg(1, 1); // 每个网格下的线程块个数(block num. per grid)
                      // bpg(blockDim.x, blockDim.y)
    dim3 tpb(N, N*N); // 每个线程块下的线程个数(thread num. per block)
                    // tpb(threadIdx.x, threadIdx.y)
    // 根据资源分配的参数
    // 在GPU上执行核函数(__global__修饰的函数)
    matrixTranspose<<<bpg, tpb>>>(Ah, Bd, rowElemNumInA, colElemNumInA);


	// 从GPU端拷贝矩阵Bd到CPU端Bh
	hipMemcpy(Bh, Bd, size, hipMemcpyDeviceToHost);

	// 打印矩阵Bh
	printf("\n");
	matrixPrint(Bh, rowElemNumInB, colElemNumInB);


	// 打印向量A和B
	printf("vector A:\n");
	vectorPrint(Ah, rowElemNumInA*colElemNumInA);
	printf("vector B:\n");
	vectorPrint(Bh, rowElemNumInB*colElemNumInB);


	// 释放CPU和GPU空间
	free(Ah);
	free(Bh);
	hipFree(Ad);
	hipFree(Bd);

	return 0;
}
