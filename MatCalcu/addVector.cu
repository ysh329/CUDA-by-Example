#include	  "hip/hip_runtime.h"
#define		  N			  1000

__device__	  float		  A[N][N];
__device__	  float		  B[N][N];
__device__    float       C[N][N];

__global__ void vectorAdd(float A[N][N], float B[N][N], float C[N][N])
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	C[i][j] = A[i][j] + B[i][j];
}

int main()
{
	int bpg = 1;
	dim3 tpb(N, N);
	vectorAdd<<<bpg, tpb>>>(A, B, C);

	hipFree(A);
	hipFree(B);
	hipFree(C);
	return 0;
}
